#include "hip/hip_runtime.h"
/*
 * Demonstration of an osgPPU Module which using a CUDA-Kernel to blur the input texture.
 */

#ifndef __KERNEL_H_
#define __KERNEL_H_


#define _DEBUG

// here workaround for broken CUDA compiler (should be solved with CUDA 2.2)
using namespace std;

#include <osgPPU/UnitInOutModule.h>
#include <cuda_gl_interop.h>
#include "cutil.h"

using namespace osgPPU;


//-----------------------------------------------------------------------------
// clamp x to range [a, b]
// GPU code
//-----------------------------------------------------------------------------
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

//-----------------------------------------------------------------------------
// get pixel from 2D image, with clamping to border
// GPU code
//-----------------------------------------------------------------------------
__device__ float4 getPixel(float4 *data, int x, int y, int width, int height)
{
    x = clamp(x, 0, width-1);
    y = clamp(y, 0, height-1);
    return data[y*width+x];
}

// macros to make indexing shared memory easier
#define SMEM(X, Y) sdata[(Y)*tilew+(X)]

//-----------------------------------------------------------------------------
// CUDA kernel to do a simple blurring
//-----------------------------------------------------------------------------
__global__ void blurKernel(float4* g_data, float4* g_odata, int imgw, int imgh, int tilew, int r, float threshold, float highlight)
{
    extern __shared__ float4 sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

    // copy tile to shared memory
    // center region
    SMEM(r + tx, r + ty) = getPixel(g_data, x, y, imgw, imgh);

    // borders
    if (threadIdx.x < r) {
        // left
        SMEM(tx, r + ty) = getPixel(g_data, x - r, y, imgw, imgh);
        // right
        SMEM(r + bw + tx, r + ty) = getPixel(g_data, x + bw, y, imgw, imgh);
    }
    if (threadIdx.y < r) {
        // top
        SMEM(r + tx, ty) = getPixel(g_data, x, y - r, imgw, imgh);
        // bottom
        SMEM(r + tx, r + bh + ty) = getPixel(g_data, x, y + bh, imgw, imgh);
    }

    // load corners
    if ((threadIdx.x < r) && (threadIdx.y < r)) {
        // tl
        SMEM(tx, ty) = getPixel(g_data, x - r, y - r, imgw, imgh);
        // bl
        SMEM(tx, r + bh + ty) = getPixel(g_data, x - r, y + bh, imgw, imgh);
        // tr
        SMEM(r + bw + tx, ty) = getPixel(g_data, x + bh, y - r, imgw, imgh);
        // br
        SMEM(r + bw + tx, r + bh + ty) = getPixel(g_data, x + bw, y + bh, imgw, imgh);
    }

    // wait for loads to complete
    __syncthreads();

    // perform convolution
    float samples = 0.0;
    float3 pixelSum = make_float3(0,0,0);

    for(int dy=-r; dy<=r; dy++) {
        for(int dx=-r; dx<=r; dx++) {
#if 0
            // try this to see the benefit of using shared memory
            float4 pixel = getPixel(g_data, x+dx, y+dy, imgw, imgh);
#else
            float4 pixel = SMEM(r+tx+dx, r+ty+dy);
#endif
            // only sum pixels within disc-shaped kernel
            float l = dx*dx + dy*dy;
            if (l <= r*r)
            {
                pixelSum.x += pixel.x;
                pixelSum.y += pixel.y;
                pixelSum.z += pixel.z;
                samples += 1.0;
            }
        }
    }

    // normalize
    pixelSum.x /= samples;
    pixelSum.y /= samples;
    pixelSum.z /= samples;

    g_odata[y*imgw+x] = make_float4(pixelSum.x, pixelSum.y, pixelSum.z, 1.0);
}

//-----------------------------------------------------------------------------
// Class capable of rendering the stuff through cuda
//-----------------------------------------------------------------------------
class ProcessingModule : public UnitInOutModule::Module
{
    public:
        ProcessingModule(UnitInOutModule* parent) : UnitInOutModule::Module(parent)
        {
            // to get all thing properly we have to specify one input and one output pbo
            parent->setUsePBOForInputTexture(0);
            parent->setUsePBOForOutputTexture(0);


            osg::notify(osg::INFO) << "osgPPU - Module - cudaKernel initialize" << std::endl;
        }

        ~ProcessingModule()
        {
            // force exit of CUDA
            hipDeviceSynchronize();
            hipDeviceReset();
            osg::notify(osg::INFO) << "osgPPU - Module - cudaKernel release" << std::endl;
        }

        //-----------------------------------------------------------------------------
        // Initialize cuda environment
        //-----------------------------------------------------------------------------
        bool init()
        {
            int deviceCount;                                                         
            CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));                
            if (deviceCount == 0) {                                                  
                fprintf(stderr, "error: no devices supporting CUDA.\n");       
                exit(EXIT_FAILURE);                                                  
            }                                                                        
            int dev = 0;                                                             
            if (dev > deviceCount-1) dev = deviceCount - 1;                          
            hipDeviceProp_t deviceProp;                                               
            CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));       
            if (deviceProp.major < 1) {                                              
                fprintf(stderr, "error: device does not support CUDA.\n");     
                exit(EXIT_FAILURE);                                                  
            }                                                                        
            CUDA_SAFE_CALL(hipSetDevice(dev));

            // print some debug info
            printf("Cuda BlurKernel Module for osgPPU:\n");
            printf("\tDevice: %s\n", deviceProp.name);
            printf("\tTotal Memory: %d MB\n", deviceProp.totalGlobalMem/1000000);
            printf("\tClock Rate: %d MHz\n", deviceProp.clockRate/1000);
            printf("\tMultiprocessors: %d\n", deviceProp.multiProcessorCount);
            printf("\tShared Mem per Block: %d\n", deviceProp.sharedMemPerBlock);
            printf("\tMax Threads per Block: %d\n", deviceProp.maxThreadsPerBlock);
            printf("\tCompute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);

            return true;
        }


        //-----------------------------------------------------------------------------
        // Register/Map textures into CUDA space and process them
        //-----------------------------------------------------------------------------
        bool beginAndProcess()
        {
            // get first input pbo
            const osg::PixelDataBufferObject* ipbo = _parent->getInputPBO(0);
            const osg::PixelDataBufferObject* opbo = _parent->getOutputPBO(0);
            if (ipbo == NULL || opbo == NULL) return false;

            // get dimensions of the input data
            int width = _parent->getViewport() ? (int)_parent->getViewport()->width() : 0;
            int height = _parent->getViewport() ? (int)_parent->getViewport()->height() : 0;
            if (width == 0 || height == 0) return false;

            // map input data
            float4* in_data = NULL;
            CUDA_SAFE_CALL(cudaGLRegisterBufferObject(ipbo->buffer(0)));
            CUDA_SAFE_CALL(cudaGLMapBufferObject( (void**)&in_data, ipbo->buffer(0)));

            // map output data
            float4* out_data = NULL;
            CUDA_SAFE_CALL(cudaGLRegisterBufferObject(opbo->buffer(0)));
            CUDA_SAFE_CALL(cudaGLMapBufferObject( (void**)&out_data, opbo->buffer(0)));


            //-----------------------------------------------------------------------------
            // run CUDA Kernel on the loaded data

            // radius of how much sampling points around the current one        
            int radius = 4;

            // specifies the number of threads per block (here 16*16*1=256 threads)
            dim3 block(16, 16, 1);

            // specifies the number of blocks used
            dim3 grid(width / block.x, height / block.y, 1);

            // size of the shared memory (memory used by each block) reflects the size of sampling points around
            int sbytes = (block.x+(2*radius))*(block.y+(2*radius)) * sizeof(float4);
        
            // run kernel with the specified parameters
            blurKernel<<< grid, block, sbytes>>>(in_data, out_data, width, height, block.x+(2*radius), radius, 0.8f, 4.0f);

            //-----------------------------------------------------------------------------


            // don't render anything afterwards
            return false;
        }


        //-----------------------------------------------------------------------------
        // Unmap/Unregister data, so that results are copied back
        //-----------------------------------------------------------------------------
        void end()
        {
            // get first input pbo
            const osg::PixelDataBufferObject* ipbo = _parent->getInputPBO(0);
            const osg::PixelDataBufferObject* opbo = _parent->getOutputPBO(0);
            if (ipbo == NULL || opbo == NULL) return;

            // unmap and unregister input data
            CUDA_SAFE_CALL(cudaGLUnmapBufferObject(opbo->buffer(0)));
            CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(opbo->buffer(0)));

            // map output data
            CUDA_SAFE_CALL(cudaGLUnmapBufferObject(ipbo->buffer(0)));
            CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(ipbo->buffer(0)));
        }

};


//-----------------------------------------------------------------------------
osg::ref_ptr<ProcessingModule> g_ProcessingModule;

//-----------------------------------------------------------------------------
// Register the module by the corresponding UnitInOutModule, so that it can be
// used in former processing operations.
//-----------------------------------------------------------------------------
extern "C" bool OSGPPU_MODULE_ENTRY(UnitInOutModule* parent) 
{
    g_ProcessingModule = new ProcessingModule(parent);
    parent->setModule(g_ProcessingModule.get());

    return true;
}


#endif 

